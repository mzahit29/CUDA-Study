#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Examples.cuh"
#include <>


__global__ void hello_cuda()
{
	printf("Hello Cuda\n");
}

__global__ void print_thread_id()
{
	printf("Hello Cuda tid[%d, %d, %d]\n",
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y,
		threadIdx.z + blockDim.z * blockIdx.z);
		//threadIdx.x, threadIdx.y, threadIdx.z);
}
