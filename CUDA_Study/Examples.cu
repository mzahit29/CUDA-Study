#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Examples.cuh"
#include <>


__global__ void hello_cuda()
{
	printf("Hello Cuda\n");
}

__global__ void print_thread_id()
{
	printf("Hello Cuda tid[%d, %d, %d]\n",
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y,
		threadIdx.z + blockDim.z * blockIdx.z);
		//threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_thread_variables()
{
	printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, BlockDim{%d,%d,%d}, GridDim{%d,%d,%d}\n",
		threadIdx.x, threadIdx.y, threadIdx.z,
		blockIdx.x, blockIdx.y, blockIdx.z,
		blockDim.x, blockDim.y, blockDim.z,
		gridDim.x, gridDim.y, gridDim.z
		);
}

__global__ void print_unique_thread_id_1D()
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, tid{%d}\n",
		threadIdx.x, threadIdx.y, threadIdx.z,
		blockIdx.x, blockIdx.y, blockIdx.z,
		tid);
}

__global__ void print_unique_thread_id_3D(int * data)
{
	int thread_count_in_block = blockDim.x * blockDim.y * blockDim.z;
	
	// Inside a block, threads access consecutive elements of array
	// (for z=0) 0 1  (for z=1) 4 5
	//           2 3            6 7
	int tid_in_block = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;

	int bid_in_grid = gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

	
	int gid = bid_in_grid * thread_count_in_block + tid_in_block;

	printf("gid{%d}, data[%d] = %d\n", gid, gid, data[gid]);
}
