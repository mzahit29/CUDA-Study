#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Examples.cuh"
#include <>


__global__ void hello_cuda()
{
	printf("Hello Cuda\n");
}

__global__ void print_thread_id()
{
	printf("Hello Cuda tid[%d, %d, %d]\n",
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y,
		threadIdx.z + blockDim.z * blockIdx.z);
		//threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_thread_variables()
{
	printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, BlockDim{%d,%d,%d}, GridDim{%d,%d,%d}\n",
		threadIdx.x, threadIdx.y, threadIdx.z,
		blockIdx.x, blockIdx.y, blockIdx.z,
		blockDim.x, blockDim.y, blockDim.z,
		gridDim.x, gridDim.y, gridDim.z
		);
}

__global__ void print_unique_thread_id_1D()
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, tid{%d}\n",
		threadIdx.x, threadIdx.y, threadIdx.z,
		blockIdx.x, blockIdx.y, blockIdx.z,
		tid);
}
