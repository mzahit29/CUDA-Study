#include "hip/hip_runtime.h"
#include ""

#include "Examples.cuh"


int main()
{
  
	int nx{16};
	int ny{1};
	dim3 block(8, 1, 1);
	dim3 grid(nx / block.x, ny / block.y, 1);

	print_unique_thread_id_1D << < grid, block>> > ();

	hipDeviceSynchronize();

	hipDeviceReset();

    return 0;
}