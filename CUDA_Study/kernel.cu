#include "hip/hip_runtime.h"
#include ""

#include "Examples.cuh"


int main()
{
  
	int nx{16};
	int ny{4};
	dim3 block(8, 2, 1);
	dim3 grid(nx / block.x, ny / block.y, 1);

	print_thread_variables << < grid, block>> > ();

	hipDeviceSynchronize();

	hipDeviceReset();

    return 0;
}