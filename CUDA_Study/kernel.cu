#include "hip/hip_runtime.h"
#include ""

#include "Examples.cuh"
#include <cstdlib>
#include <time.h>
#include <stdio.h>


int main()
{
  
	int nx{ 4 };
	int ny{ 4 };
	int nz{ 4 };

	dim3 block(2, 2, 2);
	dim3 grid(nx / block.x, ny / block.y, nz / block.z);


	// Allocate memory for 64 ints on host
	int array_size{ 64 };
	int * data_h = (int *)malloc(array_size * sizeof(int));

	// Fill the array with random values between 0-63
	srand(time(0));
	for (int i = 0; i < array_size; ++i)
	{
		data_h[i] = rand() % array_size;
	}

	// Print the array contents for debugging
	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", data_h[i]);
	} printf("\n");

	// Allocate memory on device
	int * data_d;
	hipMalloc((void **)&data_d, array_size * sizeof(int));

	// Copy the host array 
	hipMemcpy(data_d, data_h, array_size * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice);

	print_unique_thread_id_3D << < grid, block>> > (data_d);

	// Wait for GPU threads to finish
	hipDeviceSynchronize();

	// Free the allocated memory on host and device
	free(data_h);
	hipFree(data_d);

	hipDeviceReset();

    return 0;
}
