#include "hip/hip_runtime.h"
#include ""

#include "Examples.cuh"


int main()
{
  
	dim3 block(2, 1, 1);  // 2 threads in x dimension per block
	dim3 grid(4, 1, 1);  // 4 blocks in x dimension

	hello_cuda << <grid, block >> > ();

	hipDeviceSynchronize();

	hipDeviceReset();

    return 0;
}